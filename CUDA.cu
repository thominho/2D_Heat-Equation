#include<stdio.h>
#include<stdlib.h>
#include<sys/types.h>
#include<sys/stat.h>
#include<fcntl.h>
#include<unistd.h>
#include<errno.h>
#include<hip/hip_runtime.h>




#define NXPROB      288                /* x dimension of problem grid */
#define NYPROB      288                 /* y dimension of problem grid */
#define STEPS      	100              /* number of time steps */
#define MAXWORKER   16                  /* maximum number of worker tasks */
#define MINWORKER   1                  /* minimum number of worker tasks */



__global__ void updateU(float * u, int time)
{

	/*i - row and j - column*/
	int i = (blockIdx.x*blockDim.x + threadIdx.x);
	int j = (blockIdx.y*blockDim.y + threadIdx.y);
	
	if (i >= NXPROB || j >= NYPROB) return;
	
	if (i%NXPROB == 0 || j%NYPROB == 0 || j%NYPROB == NYPROB - 1 || i%NXPROB == NXPROB - 1) return;

	
	int offsetw = (time % 2 == 0) ? NXPROB*NYPROB : 0;
	int offsetr = (offsetw == 0) ? NYPROB*NXPROB : 0;

	u[offsetw + i*NYPROB + j] = u[offsetr + i*NYPROB + j] +

		0.1 * (u[offsetr + i*NYPROB + NYPROB + j] + u[offsetr + i*NYPROB - NYPROB + j] - 2 * u[offsetr + i*NYPROB + j])

		+ 0.1 * (u[offsetr + i*NYPROB + j + NYPROB] + u[offsetr + i*NYPROB + j - NYPROB] - 2 * u[offsetr + i*NYPROB + j]);

}


int main(int argc, char* argv[])
{

	printf("Cuda-Start\n");

	hipDeviceProp_t deviceProp;
	hipError_t error;
	hipGetDeviceProperties(&deviceProp, 0);
	int sum_mem = 2 * NYPROB*NXPROB;
	hipEvent_t start, stop;
	float time = 0;
	int i;


	float * u = (float *)malloc(sizeof(float)*sum_mem);
	float * device_u;
	

	for (i = 0; i<sum_mem / 2; i++){
		if ((i%NXPROB == 0) || (i%NYPROB == NYPROB - 1)){
			u[i] = 0;
			continue;
		}
		if (i < NXPROB){
			u[i] = 0;
			continue;
		}
		/*if (i == NXPROB){
			u[i] == 0.0;
			continue;
		}*/
		if ((i<NYPROB*NXPROB) && (i>(NXPROB-1)*NYPROB)){
			u[i] = 0;
			continue;
		}
		//printf("WILL WRITE TO %d\n", i);
		u[i] = rand() % 100;
	}

	for (i = sum_mem / 2; i < sum_mem; i++){
		u[i] = 0;
	}

	error = hipMalloc(&device_u, sum_mem*sizeof(float));
	
	if (error != hipSuccess){ fprintf(stderr, "Failed to allocate memory for matrix  %s\n", hipGetErrorString(error)); return -4; }
	
	error = hipMemcpy(device_u, u, sum_mem*sizeof(float), hipMemcpyHostToDevice);

	if (error != hipSuccess){ fprintf(stderr, "Failed to copy matrix to device: %s\n", hipGetErrorString(error)); fflush(stderr); return -7; }

	int root;
	for (root = 2; root*root <= deviceProp.maxThreadsPerBlock; root++)
		if (root*root == deviceProp.maxThreadsPerBlock) break;

	if (root*root>deviceProp.maxThreadsPerBlock) root--;

	error = hipEventCreate(&start);
	if (error != hipSuccess){ fprintf(stderr, "Failure(time): %s\n", hipGetErrorString(error));fflush(stderr); return-17; }

	error = hipEventCreate(&stop);
	if (error != hipSuccess){ fprintf(stderr, "Failure(time): %s\n", hipGetErrorString(error));fflush(stderr); return-17; }

	dim3 threadsPerBlock(root, root);
	int blockDimX = (NXPROB%root==0) ? (NXPROB / root) : (NXPROB / root + 1);
	int blockDimY = (NYPROB%root==0) ? (NYPROB / root) : (NYPROB / root + 1);
	dim3 numOfBlocks(blockDimX, blockDimY, 1);//pixsize
	hipEventRecord(start);

	for (i = 0; i<STEPS; i++){
		updateU <<< numOfBlocks, threadsPerBlock>>>(device_u,i);//3o shared
		error = hipGetLastError();
		if (error != hipSuccess){ fprintf(stderr, "Error in steps call %s\n", hipGetErrorString(error));fflush(stderr); return -11; }
		
	}

	hipEventRecord(stop);
	error = hipMemcpy(u, device_u, sum_mem*sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess){ fprintf(stderr, "Failed to copy matrix(to host): %s\n", hipGetErrorString(error));fflush(stderr); return -8; }

	hipEventElapsedTime(&time, start, stop);
	printf("Time %f \n", time);
	
	return 0;
}
